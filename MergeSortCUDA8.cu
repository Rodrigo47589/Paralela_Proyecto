#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <hip/hip_runtime.h>

// Ajusta cu�ntos hilos tendr� cada bloque CUDA
#define BLOCK_SIZE 1024

// Funci�n para obtener la cantidad de n�cleos CUDA de la GPU activa
int getCudaCores() {
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    // Cada arquitectura NVIDIA tiene diferente n�mero de n�cleos por multiprocesador
    int cores_per_sm = 0;
    switch (prop.major) {
    case 7: // Turing (GTX 1650)
        cores_per_sm = 64;
        break;
    }
    return prop.multiProcessorCount * cores_per_sm;
}

// Kernel CUDA: cada hilo fusiona un segmento ("run") del arreglo
__global__
void merge_kernel(const float* A, const int* idx_A, float* B, int* idx_B,
    int N, int width) {
    // Cada hilo calcula el inicio de su segmento
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int segSize = 2 * width;
    int segStart = tid * segSize;
    if (segStart >= N) return; // Si se pasa del tama�o, no hace nada

    // Definir los l�mites de los dos subarreglos a fusionar
    int mid = min(segStart + width, N);
    int end = min(segStart + segSize, N);

    // Fusi�n ordenada (descendente)
    int i = segStart, j = mid, k = segStart;
    while (i < mid && j < end) {
        if (A[i] >= A[j]) {
            B[k] = A[i];
            idx_B[k] = idx_A[i];
            i++;
        }
        else {
            B[k] = A[j];
            idx_B[k] = idx_A[j];
            j++;
        }
        k++;
    }
    // Copia remanente si queda
    while (i < mid) {
        B[k] = A[i];
        idx_B[k] = idx_A[i];
        i++; k++;
    }
    while (j < end) {
        B[k] = A[j];
        idx_B[k] = idx_A[j];
        j++; k++;
    }
}

// Funci�n que controla el merge sort en GPU y muestra info de cada pasada
void merge_sort_gpu(float* d_in, int* d_idx, float* d_out, int* d_out_idx, int N, int cuda_cores) {
    int* d_temp_idx;
    float* d_temp;
    hipMalloc(&d_temp, N * sizeof(float));
    hipMalloc(&d_temp_idx, N * sizeof(int));
    hipMemcpy(d_temp, d_in, N * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(d_temp_idx, d_idx, N * sizeof(int), hipMemcpyDeviceToDevice);

    bool flip = false;
    // La variable width define el tama�o de run a fusionar (1, 2, 4, 8, ...)
    for (int width = 1; width < N; width *= 2) {
        int numSegs = (N + 2 * width - 1) / (2 * width); // Cu�ntos segmentos en esta pasada
        int numBlocks = (numSegs + BLOCK_SIZE - 1) / BLOCK_SIZE;
        int totalHilos = numBlocks * BLOCK_SIZE;

        // Imprimir c�mo se distribuye el trabajo en GPU en cada pasada
        std::cout << "[PASADA width=" << width
            << "] segmentos=" << numSegs
            << ", bloques=" << numBlocks
            << ", hilosPorBloque=" << BLOCK_SIZE
            << ", totalHilosLanzados=" << totalHilos
            << ", CUDA_cores=" << cuda_cores
            << std::endl;

        if (!flip) {
            merge_kernel << <numBlocks, BLOCK_SIZE >> >
                (d_temp, d_temp_idx, d_out, d_out_idx, N, width);
        }
        else {
            merge_kernel << <numBlocks, BLOCK_SIZE >> >
                (d_out, d_out_idx, d_temp, d_temp_idx, N, width);
        }
        hipDeviceSynchronize();
        flip = !flip;
    }
    // Copiar al buffer final si qued� en el temporal
    if (!flip) {
        hipMemcpy(d_out, d_temp, N * sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(d_out_idx, d_temp_idx, N * sizeof(int), hipMemcpyDeviceToDevice);
    }
    hipFree(d_temp);
    hipFree(d_temp_idx);
}

// Lee el CSV, omitiendo cabecera, y llena dos vectores: nombres y scores
void leerCSV(const std::string& filename, std::vector<std::string>& nombres, std::vector<float>& scores) {
    std::ifstream in(filename);
    std::string line;
    bool primera = true;
    while (std::getline(in, line)) {
        if (primera) { primera = false; continue; } // Salta cabecera
        std::stringstream ss(line);
        std::string nombre, scoreStr;
        if (!std::getline(ss, nombre, ',')) continue;
        if (!std::getline(ss, scoreStr)) continue;
        if (nombre.empty() || scoreStr.empty()) continue;
        try {
            scores.push_back(std::stof(scoreStr));
            nombres.push_back(nombre);
        }
        catch (...) {
            std::cerr << "L�nea con formato incorrecto: " << line << std::endl;
        }
    }
}

// Escribe los resultados ordenados en un archivo CSV
void escribirCSV(const std::string& filename, const std::vector<std::string>& nombres, const std::vector<float>& scores) {
    std::ofstream out(filename);
    for (size_t i = 0; i < nombres.size(); ++i) {
        out << nombres[i] << "," << scores[i] << "\n";
    }
}

int main(int argc, char* argv[]) {
    std::string archivo_in = argc > 1 ? argv[1] : "usuarios.csv";
    std::string archivo_out = argc > 2 ? argv[2] : "usuarios_ordenados_merge.csv";
    std::vector<std::string> nombres;
    std::vector<float> scores;
    leerCSV(archivo_in, nombres, scores);
    int N = scores.size();
    std::vector<int> idx(N);
    for (int i = 0; i < N; ++i) idx[i] = i;

    float* d_in;     int* d_idx;
    float* d_out;    int* d_out_idx;
    hipMalloc(&d_in, N * sizeof(float));
    hipMalloc(&d_idx, N * sizeof(int));
    hipMalloc(&d_out, N * sizeof(float));
    hipMalloc(&d_out_idx, N * sizeof(int));
    hipMemcpy(d_in, scores.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_idx, idx.data(), N * sizeof(int), hipMemcpyHostToDevice);

    // Obtiene y muestra el n�mero de CUDA cores f�sicos de la GPU
    int cuda_cores = getCudaCores();

    // Toma el tiempo de ejecuci�n en GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    merge_sort_gpu(d_in, d_idx, d_out, d_out_idx, N, cuda_cores);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("Tiempo GPU Merge Sort: %.4f ms\n", ms);

    // Copia los resultados a memoria del host
    std::vector<float> scores_sorted(N);
    std::vector<int> idx_sorted(N);
    hipMemcpy(scores_sorted.data(), d_out, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(idx_sorted.data(), d_out_idx, N * sizeof(int), hipMemcpyDeviceToHost);

    // Muestra el top 10 en pantalla
    std::vector<std::string> nombres_sorted(N);
    for (int i = 0; i < N; ++i)
        nombres_sorted[i] = nombres[idx_sorted[i]];

    std::cout << "\n=== TOP 10 USUARIOS ===" << std::endl;
    for (int i = 0; i < std::min(N, 10); ++i) {
        std::cout << (i + 1) << ". " << nombres_sorted[i] << " - " << scores_sorted[i] << std::endl;
    }
    std::cout << "=======================\n" << std::endl;

    escribirCSV(archivo_out, nombres_sorted, scores_sorted);
    printf("Archivo ordenado guardado en: %s\n", archivo_out.c_str());

    hipFree(d_in); hipFree(d_idx); hipFree(d_out); hipFree(d_out_idx);
    return 0;
}
